#include "hip/hip_runtime.h"
#include "PCAFeatureMaps.cuh"
#include <hip/hip_runtime.h>
#include <cmath>

#define CUDA_CHECK(err) /* same as your other kernels */

// Kernel: one thread per output feature cell+channel
__global__ void pcaKernel(
    const float* __restrict__ inMap,
    float*       __restrict__ outMap,
    int sizeX, int sizeY,
    int inChans, int outChans,
    float normFactor)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int total = sizeX*sizeY*outChans;
  if (idx >= total) return;

  int chan = idx / (sizeX*sizeY);
  int cell = idx % (sizeX*sizeY);
  // sum over inChans:
  float sum = 0.0f;
  for (int k = 0; k < inChans; ++k) {
    sum += inMap[k*(sizeX*sizeY) + cell];
  }
  outMap[chan*(sizeX*sizeY) + cell] = sum * normFactor;
}

int PCAFeatureMapsGPU(CvLSVMFeatureMapCaskade *map) {
  int sizeX   = map->sizeX, sizeY = map->sizeY;
  int inChans = map->numFeatures;      // = pp from normalize step
  int outChans= /* whatever your serial code reduces to, e.g. NUM_SECTOR*4 */;
  int cells   = sizeX*sizeY;
  float normF = /* your serial code’s ny value */;

  size_t inBytes  = sizeof(float)*cells*inChans;
  size_t outBytes = sizeof(float)*cells*outChans;

  // 1) allocate & copy input if you haven’t already
  float *d_in, *d_out;
  CUDA_CHECK(hipMalloc(&d_in,  inBytes));
  CUDA_CHECK(hipMalloc(&d_out, outBytes));
  CUDA_CHECK(hipMemcpy(d_in, map->map, inBytes, hipMemcpyHostToDevice));

  // 2) kernel launch
  int threads = 256, blocks = (cells*outChans + threads-1)/threads;
  pcaKernel<<<blocks,threads>>>(d_in, d_out, sizeX, sizeY, inChans, outChans, normF);
  CUDA_CHECK(hipDeviceSynchronize());

  // 3) copy back & clean up
  float *h_out = (float*)malloc(outBytes);
  CUDA_CHECK(hipMemcpy(h_out, d_out, outBytes, hipMemcpyDeviceToHost));
  hipFree(d_in);
  hipFree(d_out);

  // 4) swap buffers & update map struct
  free(map->map);
  map->map        = h_out;
  map->numFeatures= outChans;
  return LATENT_SVM_OK;
}
