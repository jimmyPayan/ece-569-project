#define NUM_SECTOR 9
#define BLOCK_DIM 16
#define K_MAX 4
#define FEATURES_MAX 27


#include <hip/hip_runtime.h>
#include <stdio.h>


// First optimization: Proper block dimensions (32x32 -> 16x16). Preparatory optimization that resulted in -10s from singlescale and -2s from Multiscale. Done so that shared memory (optimization 2) will not be > 48 kB.
// Timing functions  give tools to directly evaluate speedups! Comment out if you want max performance, I suppose.

/*  NOTE: Clock Rate = 1328500 kHz */

/* 	UPDATE THIS COMMENT AFTER ANY AND ALL OPTIMIZATIONS PLEASE <3
	Thread 0,0 of Block 0,0 took 27266 total cycles. It required:
	~ 1022 cycles to write to shared memory.
	~ 14089 cycles to compute data.
	~ 11702 cycles to convert shared memory to global memory.
	453 cycles unaccounted for.

	IF YOU WAIT FOR ALL 240 PRINT STATEMENTS TO END YOU GET THIS TOO
	Total execution time: 16850.0 ms.

	--- Function Timing Summary ---
	Total time spent in getFeatures(): 2.0 s
	Total time spent in gaussianCorrelation(): 0.7 s
	Total time spent in train(): -1.8 s
	Total time spent in detect(): -1.5 s
	Total execution time: -0.7 s
*/


// Still using atomics excessively, so we might not see speedup at all.  No removals of if() statements
// k seems to be passed in as cell_size, which is set to 4... for loop should be okay.

__global__ void kernel_n4(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *d_map, int stringSize, int *d_alfa, float *d_r, float *d_w,  int *d_nearest) {
	long long int phase0, c_start;
	phase0 = clock64();

	// Allocate shared memory
	__shared__ float shared_w[K_MAX * 2];
	__shared__ int shared_nearest[K_MAX];
	__shared__ float shared_blockMap[BLOCK_DIM * BLOCK_DIM * FEATURES_MAX];

	// Use thread IDs as iterators, same names as joaofaro to keep me sane while debugging
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int Idx = threadIdx.x + threadIdx.y * blockDim.x;	

	int a, d;
	
	int phase1, phase2, phase3;

	int nearest_ii, nearest_jj;

	c_start = clock64();
	if (Idx < k * 2)
		shared_w [Idx] = d_w[Idx];

	if (Idx < k)
		shared_nearest[Idx] = d_nearest[Idx];

	if (Idx < (BLOCK_DIM * BLOCK_DIM)) {
		for (a = 0 ; a < numFeatures; a++) {
			shared_blockMap[Idx * numFeatures + a] = 0.0f;
		}
	}
	phase1 = (int) (clock64() - c_start);
	__syncthreads();

c_start = clock64();
if (i < sizeY && j < sizeX) {
    for (int ii = 0; ii < k; ii++) {
    for (int jj = 0; jj < k; jj++) {
        if ((i * k + ii > 0) && 
            (i * k + ii < height - 1) && 
            (j * k + jj > 0) && 
            (j * k + jj < width - 1))
        {
            d = (k * i + ii) * width + (j * k + jj);
			
			nearest_ii = shared_nearest[ii];
			nearest_jj = shared_nearest[jj];

            shared_blockMap[Idx * numFeatures + d_alfa[d * 2]] += d_r[d] * shared_w[ii * 2] * shared_w[jj * 2];

            shared_blockMap[Idx * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += d_r[d] * shared_w[ii * 2] * shared_w[jj * 2];

			
            if (((int)threadIdx.x + nearest_ii >= 0) && 
                ((int)threadIdx.x + nearest_ii < BLOCK_DIM))
            {

                shared_blockMap[((threadIdx.x + nearest_ii) + threadIdx.y * blockDim.x) * numFeatures + d_alfa[d * 2]] += d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2];

                shared_blockMap[((threadIdx.x + nearest_ii) + threadIdx.y * blockDim.x) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2];
            }
			
			
            if (((int)threadIdx.y + nearest_jj >= 0) && 
                ((int)threadIdx.y + nearest_jj < BLOCK_DIM))
            {

                shared_blockMap[(threadIdx.x + (threadIdx.y + shared_nearest[jj]) * blockDim.x) * numFeatures + d_alfa[d * 2]] += d_r[d] * shared_w[ii * 2] * shared_w[jj * 2 + 1];

                shared_blockMap[(threadIdx.x + (threadIdx.y + shared_nearest[jj]) * blockDim.x) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += d_r[d] * shared_w[ii * 2] * shared_w[jj * 2 + 1];
            }

            if (((int)threadIdx.x + nearest_ii 	>= 0) && 
                ((int)threadIdx.x + nearest_ii 	< BLOCK_DIM) &&
                ((int)threadIdx.y + nearest_jj  >= 0) && 
                ((int)threadIdx.y + nearest_jj	< BLOCK_DIM))
            {

                shared_blockMap[((threadIdx.x + nearest_ii) + (threadIdx.y + nearest_jj) * blockDim.x) * numFeatures + d_alfa[d * 2]] += d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2 + 1];

                shared_blockMap[((threadIdx.x + nearest_ii) + (threadIdx.y + nearest_jj) * blockDim.x) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2 + 1];
            }
        }
    }/*for(jj = 0; jj < k; jj++)*/
    }/*for(ii = 0; ii < k; ii++)*/
}/*if (i < sizeY && j < sizeX)*/

	__syncthreads();
	phase2 = (int) (clock64() - c_start);

	// Write to global memory
	c_start = clock64();
	for (int a = 0; a < numFeatures; a++) {
		d_map[i * stringSize + j * numFeatures + a] = shared_blockMap[Idx * numFeatures + a];
	}
	phase3 = (int) (clock64() - c_start);
	phase0 = clock64() - phase0;
	if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
		printf("Thread 0,0 of Block 0,0 took %d total cycles. It required:\n~ %d cycles to write to shared memory.\n~ %d cycles to compute data.\n~ %d cycles to convert shared memory to global memory.\n%d cycles unaccounted for.\n", (int) phase0, phase1, phase2, phase3, ((int) phase0 - (phase1 + phase2 + phase3)));	
	}

}

void featureGPU(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *map, int stringSize, int *alfa, float *r, float *w, int *nearest){
	/*
	cudaDeviceProp prop;
	cudaGetDeviceProperties(&prop, 0); // 0 = device ID (first GPU)
	printf("Clock Rate: %d kHz\n", (int)prop.clockRate);
	*/  

	float *d_map, *d_r, *d_w;
    int *d_alfa, *d_nearest;
    hipMalloc((void**) &d_map, sizeof(float) * (sizeX * sizeY * numFeatures));
    hipMalloc((void**) &d_alfa, sizeof(int) * (width * height * 2));
    hipMalloc((void**) &d_r, sizeof(float) * (width * height));
    hipMalloc((void**) &d_w, sizeof(float) * (k * 2));
    hipMalloc((void**) &d_nearest, sizeof(int) * k);

    hipMemcpy(d_map, map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyHostToDevice);
    hipMemcpy(d_alfa, alfa, sizeof(int) * (width * height * 2) , hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(float) * (width * height), hipMemcpyHostToDevice); 
    hipMemcpy(d_w, w, sizeof(float) * (k * 2), hipMemcpyHostToDevice);
    hipMemcpy(d_nearest, nearest, sizeof(int) * k, hipMemcpyHostToDevice); 

    // Total number of threads needed: sizeY * sizeX * k... max value of k is 4, which occurs during HOG. 1024 / 4 = 256, sqrt(256) = 16.
    const dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);
    const dim3 blocksPerGrid(ceil((float)sizeY / BLOCK_DIM), ceil((float)sizeX / BLOCK_DIM));

    kernel_n4<<<blocksPerGrid,threadsPerBlock>>>(sizeY, sizeX, k, height, width, numFeatures, d_map, stringSize, d_alfa, d_r, d_w, d_nearest);

    hipMemcpy(map, d_map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyDeviceToHost);

    hipFree(d_map);
    hipFree(d_alfa);
    hipFree(d_r);
    hipFree(d_w);
    hipFree(d_nearest);
}
