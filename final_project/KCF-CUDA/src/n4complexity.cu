#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define NUM_SECTOR 9

// Naive 3D Approach first. Using atomics excessively, we might not see speedup at all.  No memory optimizations yet, no removals of if() statements
// k seems to be passed in as cell_size, which is set to 4... for loop should be okay.

__global__ void kernel_n4(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *d_map, int stringSize, int *d_alfa, float *d_r, float *d_w,  int *d_nearest) {
	
	// Use thread IDs as iterators, same names as joaofaro to keep me sane while debugging
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int ii = blockIdx.z * blockDim.z + threadIdx.z;
	
	int d;
if (i < sizeY && j < sizeX) {
	for(int jj = 0; jj < k; jj++)
          {
            if ((i * k + ii > 0) && 
                (i * k + ii < height - 1) && 
                (j * k + jj > 0) && 
                (j * k + jj < width  - 1))
            {
              d = (k * i + ii) * width + (j * k + jj);
             // d_map[ i * stringSize + j * numFeatures + d_alfa[d * 2    ]] += 
             //    d_r[d] * d_w[ii * 2] * d_w[jj * 2];
	      atomicAdd(&d_map[ i * stringSize + j * numFeatures + d_alfa[d * 2    ]],
                   d_r[d] * d_w[ii * 2] * d_w[jj * 2]);
             
	     // d_map[ i * stringSize + j * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
             //    d_r[d] * d_w[ii * 2] * d_w[jj * 2];
              atomicAdd(&d_map[ i * stringSize + j * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR],
	           d_r[d] * d_w[ii * 2] * d_w[jj * 2]);
	      
	      if ((i + d_nearest[ii] >= 0) && 
                  (i + d_nearest[ii] <= sizeY - 1))
              {
             //   d_map[(i + d_nearest[ii]) * stringSize + j * numFeatures + d_alfa[d * 2    ]             ] += 
             //   d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 ];
	      atomicAdd(&d_map[(i + d_nearest[ii]) * stringSize + j * numFeatures + d_alfa[d * 2    ],
	          d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 ]);

             //   d_map[(i + d_nearest[ii]) * stringSize + j * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
             //     d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 ];
	      atomicAdd(&d_map[(i + d_nearest[ii]) * stringSize + j * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR],
	            d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 ]);

              }
              if ((j + d_nearest[jj] >= 0) && 
                  (j + d_nearest[jj] <= sizeX - 1))
              {
             //   d_map[i * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2    ]             ] += 
             //     d_r[d] * d_w[ii * 2] * d_w[jj * 2 + 1];
	      atomicAdd(&d_map[i * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2    ]             ],
	            d_r[d] * d_w[ii * 2] * d_w[jj * 2 + 1]);          

	     //   d_map[i * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
             //     d_r[d] * d_w[ii * 2] * d_w[jj * 2 + 1];
	      atomicAdd(&d_map[i * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR],
	            d_r[d] * d_w[ii * 2] * d_w[jj * 2 + 1]);
              }
              if ((i + d_nearest[ii] >= 0) && 
                  (i + d_nearest[ii] <= sizeY - 1) && 
                  (j + d_nearest[jj] >= 0) && 
                  (j + d_nearest[jj] <= sizeX - 1))
              {
             //   d_map[(i + d_nearest[ii]) * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2    ]             ] += 
             //     d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 + 1];
	      atomicAdd(&d_map[(i + d_nearest[ii]) * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2    ]],
	            d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 + 1]);

             //   d_map[(i + d_nearest[ii]) * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
             //     d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 + 1];
	      atomicAdd(&d_map[(i + d_nearest[ii]) * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR],
	            d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 + 1]);
              }
            } // if()
} // for(int jj = 0; jj < k; jj++)
} // if (i < sizeY && j < sizeX)
}
