#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#define NUM_SECTOR 9

// Naive 3D Approach first. Can do pseudo-4D if needed. No memory optimizations yet, no removals of if() statements
// k seems to be passed in as cell_size, which is set to 4... for loop should be okay.

__global__ void kernel_n4(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *map, int stringSize, int *alfa, float *r, float *w,  int *nearest) {
	
	// Use thread IDs as iterators, same names as joaofaro to keep me sane while debugging
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int ii = blockIdx.z * blockDim.z + threadIdx.z;
	
	int d;
if (i < sizeY && j < sizeX) {
	for(int jj = 0; jj < k; jj++)
          {
            if ((i * k + ii > 0) && 
                (i * k + ii < height - 1) && 
                (j * k + jj > 0) && 
                (j * k + jj < width  - 1))
            {
              d = (k * i + ii) * width + (j * k + jj);
              map[ i * stringSize + j * numFeatures + alfa[d * 2    ]] += 
                  r[d] * w[ii * 2] * w[jj * 2];
              map[ i * stringSize + j * numFeatures + alfa[d * 2 + 1] + NUM_SECTOR] += 
                  r[d] * w[ii * 2] * w[jj * 2];
              if ((i + nearest[ii] >= 0) && 
                  (i + nearest[ii] <= sizeY - 1))
              {
                map[(i + nearest[ii]) * stringSize + j * numFeatures + alfa[d * 2    ]             ] += 
                  r[d] * w[ii * 2 + 1] * w[jj * 2 ];
                map[(i + nearest[ii]) * stringSize + j * numFeatures + alfa[d * 2 + 1] + NUM_SECTOR] += 
                  r[d] * w[ii * 2 + 1] * w[jj * 2 ];
              }
              if ((j + nearest[jj] >= 0) && 
                  (j + nearest[jj] <= sizeX - 1))
              {
                map[i * stringSize + (j + nearest[jj]) * numFeatures + alfa[d * 2    ]             ] += 
                  r[d] * w[ii * 2] * w[jj * 2 + 1];
                map[i * stringSize + (j + nearest[jj]) * numFeatures + alfa[d * 2 + 1] + NUM_SECTOR] += 
                  r[d] * w[ii * 2] * w[jj * 2 + 1];
              }
              if ((i + nearest[ii] >= 0) && 
                  (i + nearest[ii] <= sizeY - 1) && 
                  (j + nearest[jj] >= 0) && 
                  (j + nearest[jj] <= sizeX - 1))
              {
                map[(i + nearest[ii]) * stringSize + (j + nearest[jj]) * numFeatures + alfa[d * 2    ]             ] += 
                  r[d] * w[ii * 2 + 1] * w[jj * 2 + 1];
                map[(i + nearest[ii]) * stringSize + (j + nearest[jj]) * numFeatures + alfa[d * 2 + 1] + NUM_SECTOR] += 
                  r[d] * w[ii * 2 + 1] * w[jj * 2 + 1];
              }
            } // if()
} // for(int jj = 0; jj < k; jj++)
} // if (i < sizeY && j < sizeX)
}
