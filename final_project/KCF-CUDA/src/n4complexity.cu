#define NUM_SECTOR 9

#include <hip/hip_runtime.h>
#include <stdio.h>

// Naive 3D Approach first. Using atomics excessively, so we might not see speedup at all.  No memory optimizations yet, no removals of if() statements
// k seems to be passed in as cell_size, which is set to 4... for loop should be okay.

__global__ void kernel_n4(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *d_map, int stringSize, int *d_alfa, float *d_r, float *d_w,  int *d_nearest) {
	
	// Use thread IDs as iterators, same names as joaofaro to keep me sane while debugging
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	//int ii = blockIdx.z * blockDim.z + threadIdx.z;
	
	int d;
if (i < sizeY && j < sizeX) {
	for(int ii = 0; ii < k; ii++) { /////////////////////////////////////
	for(int jj = 0; jj < k; jj++)
          {
            if ((i * k + ii > 0) && 
                (i * k + ii < height - 1) && 
                (j * k + jj > 0) && 
                (j * k + jj < width  - 1))
            {
            d = (k * i + ii) * width + (j * k + jj);
           //  d_map[ i * stringSize + j * numFeatures + d_alfa[d * 2    ]] += 
           //     d_r[d] * d_w[ii * 2] * d_w[jj * 2];
	      atomicAdd(&d_map[ i * stringSize + j * numFeatures + d_alfa[d * 2    ]],
                   d_r[d] * d_w[ii * 2] * d_w[jj * 2]);
             
	   //   d_map[ i * stringSize + j * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
           //      d_r[d] * d_w[ii * 2] * d_w[jj * 2];
              atomicAdd(&d_map[ i * stringSize + j * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR],
	           d_r[d] * d_w[ii * 2] * d_w[jj * 2]);
	      
	      if ((i + d_nearest[ii] >= 0) && 
                  (i + d_nearest[ii] <= sizeY - 1))
              {
           //     d_map[(i + d_nearest[ii]) * stringSize + j * numFeatures + d_alfa[d * 2    ]             ] += 
           //     d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 ];
	      atomicAdd(&d_map[(i + d_nearest[ii]) * stringSize + j * numFeatures + d_alfa[d * 2    ]],
	          d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 ]);

           //     d_map[(i + d_nearest[ii]) * stringSize + j * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
           //       d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 ];
	      atomicAdd(&d_map[(i + d_nearest[ii]) * stringSize + j * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR],
	            d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 ]);

              }
              if ((j + d_nearest[jj] >= 0) && 
                  (j + d_nearest[jj] <= sizeX - 1))
              {
            //    d_map[i * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2    ]             ] += 
            //      d_r[d] * d_w[ii * 2] * d_w[jj * 2 + 1];
	      atomicAdd(&d_map[i * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2    ]             ],
	            d_r[d] * d_w[ii * 2] * d_w[jj * 2 + 1]);          

	    //    d_map[i * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
            //      d_r[d] * d_w[ii * 2] * d_w[jj * 2 + 1];
	      atomicAdd(&d_map[i * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR],
	            d_r[d] * d_w[ii * 2] * d_w[jj * 2 + 1]);
              }
              if ((i + d_nearest[ii] >= 0) && 
                  (i + d_nearest[ii] <= sizeY - 1) && 
                  (j + d_nearest[jj] >= 0) && 
                  (j + d_nearest[jj] <= sizeX - 1))
              {
            //    d_map[(i + d_nearest[ii]) * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2    ]             ] += 
            //      d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 + 1];
	      atomicAdd(&d_map[(i + d_nearest[ii]) * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2    ]],
	            d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 + 1]);

            //    d_map[(i + d_nearest[ii]) * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
            //      d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 + 1];
	      atomicAdd(&d_map[(i + d_nearest[ii]) * stringSize + (j + d_nearest[jj]) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR],
	            d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 + 1]);
              }
            } // if()
} // for(int jj = 0; jj < k; jj++)
} // for(int ii = 0; ii < k; ii++)
} // if (i < sizeY && j < sizeX)
}


void featureGPU(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *map, int stringSize, int *alfa, float *r, float *w, int *nearest){
	printf("Running kernel_n4.\n");
    float *d_map, *d_r, *d_w;
    int *d_alfa, *d_nearest;
// Commented out cudaMalloc()'s are passed by value
    //cudaMalloc((void**) &sizeY, sizeof(int));
    //cudaMalloc((void**) &sizeX, sizeof(int));
    //cudaMalloc((void**) &k,     sizeof(int));
    //cudaMalloc((void**) &height, sizeof(int);
    //cudaMalloc((void**) &width, sizeof(int));
    //cudaMalloc((void**) &numFeatures, sizeof(int));
    hipMalloc((void**) &d_map, sizeof(float) * (sizeX * sizeY * numFeatures));
    //cudaMalloc((void**) &stringSize, sizeof(int));
    hipMalloc((void**) &d_alfa, sizeof(int) * (width * height * 2));
    hipMalloc((void**) &d_r, sizeof(float) * (width * height));
    hipMalloc((void**) &d_w, sizeof(float) * (k * 2));
    hipMalloc((void**) &d_nearest, sizeof(int) * k);

    hipMemcpy(d_map, map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyHostToDevice);
    hipMemcpy(d_alfa, alfa, sizeof(int) * (width * height * 2) , hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(float) * (width * height), hipMemcpyHostToDevice); 
    hipMemcpy(d_w, w, sizeof(float) * (k * 2), hipMemcpyHostToDevice);
    hipMemcpy(d_nearest, nearest, sizeof(int) * k, hipMemcpyHostToDevice); 

    // Total number of threads needed: sizeY * sizeX * k... max value of k is 4, which occurs during HOG. 1024 / 4 = 256, sqrt(256) = 16.
    const dim3 threadsPerBlock(32,32);
    const dim3 blocksPerGrid(ceil((float)sizeY / 32), ceil((float)sizeX / 32));

    kernel_n4<<<blocksPerGrid,threadsPerBlock>>>(sizeY, sizeX, k, height, width, numFeatures, map, stringSize, alfa, r, w, nearest);

    hipMemcpy(map, d_map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyDeviceToHost);
    hipMemcpy(alfa, d_alfa, sizeof(int) * (width * height * 2) , hipMemcpyDeviceToHost);
    hipMemcpy(r, d_r, sizeof(float) * (width * height), hipMemcpyDeviceToHost); 
    hipMemcpy(w, d_w, sizeof(float) * (k * 2), hipMemcpyDeviceToHost);
    hipMemcpy(nearest, d_nearest, sizeof(int) * k, hipMemcpyDeviceToHost); 

    hipFree(d_map);
    hipFree(d_alfa);
    hipFree(d_r);
    hipFree(d_w);
    hipFree(d_nearest);
}
