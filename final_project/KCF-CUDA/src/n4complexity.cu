#define NUM_SECTOR 9
#define BLOCK_X 6
#define BLOCK_Y 6
#define BLOCK_Z 27
#define K_MAX 4
#define FEATURES_MAX 27


#include <hip/hip_runtime.h>
#include <stdio.h>

// Timing functions give tools to directly evaluate speedups! Comment out if you want max performance, I suppose.

/*  NOTE: Clock Rate = 1328500 kHz */

/* 	UPDATE THIS COMMENT AFTER ANY AND ALL OPTIMIZATIONS PLEASE <3 
	As a convention, just use the last thread measurement and copy alongside the Function Timing Summary
		

	OPTIMIZATION NAME: 6x6x27 Fully Active Threads with Privatization and optimized reads

	SINGLESCALE
	Thread 0,0,0 of Block 0,0 took 23402 total cycles. It required:
	~ 1139 cycles to write to shared memory.
	~ 21108 cycles to compute data.
	~ 431 cycles to convert shared memory to global memory.
	724 cycles unaccounted for.
	Total execution time: 96370.0 ms.

	--- Function Timing Summary ---
	Total time spent in getFeatures():24.5 s
	Total time spent in gaussianCorrelation(): 63.5 s
	Total time spent in train(): 7.9 s
	Total time spent in detect(): 8.0 s
	Total time spent in getFeatureMaps():16.8 s

*/

__global__ void kernel_n4(int sizeY, int sizeX, int k, int height, int width, int numFeatures,
                          float *d_map, int stringSize, int *d_alfa, float *d_r, float *d_w, int *d_nearest)
{
    long long int phase0 = clock64();

    __shared__ float shared_w[K_MAX * 2];
    __shared__ int shared_nearest[K_MAX];

    int i = blockIdx.y * BLOCK_Y + threadIdx.y;
    int j = blockIdx.x * BLOCK_X + threadIdx.x;
    int f = threadIdx.z;  // feature index (0..26)

    int phase1, phase2, phase3;

    int d;
    int nearest_ii, nearest_jj;
    int d_alfa_0, d_alfa_1;
    float w_ii_0, w_ii_1, w_jj_0, w_jj_1;
    float d_r_d;
    float acc = 0.0f;

    // Phase 1 Start: Initialize shared memory
    phase1 = clock64();
/* giving errors
    if (threadIdx.y == 0 && threadIdx.z == 0 && threadIdx.x < k * 2) {
        shared_w[threadIdx.x] = d_w[threadIdx.x];
    }
    if (threadIdx.y == 0 && threadIdx.z == 0 && threadIdx.x < k) {
        shared_nearest[threadIdx.x] = d_nearest[threadIdx.x];
    }
*/

if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    for (int t = 0; t < k * 2; ++t) {
        shared_w[t] = d_w[t];
    }
    for (int t = 0; t < k; ++t) {
        shared_nearest[t] = d_nearest[t];
    }
}

//__syncthreads();
    __syncthreads();
    phase1 = (int)(clock64() - phase1);

    // Phase 2 Start: Compute
    phase2 = clock64();

    if (i < sizeY && j < sizeX && f < numFeatures) {
        for (int ii = 0; ii < k; ii++) {
            nearest_ii = shared_nearest[ii];
            w_ii_0 = shared_w[ii * 2];
            w_ii_1 = shared_w[ii * 2 + 1];

            for (int jj = 0; jj < k; jj++) {
                int y = i * k + ii;
                int x = j * k + jj;

                if (y > 0 && y < height - 1 && x > 0 && x < width - 1) {
                    d = y * width + x;

                    d_alfa_0 = d_alfa[d * 2];
                    d_alfa_1 = d_alfa[d * 2 + 1];

                    if (d_alfa_0 >= numFeatures || d_alfa_1 + NUM_SECTOR >= numFeatures) continue;  //FIX

                    nearest_jj = shared_nearest[jj];
                    w_jj_0 = shared_w[jj * 2];
                    w_jj_1 = shared_w[jj * 2 + 1];
                    d_r_d = d_r[d];

                    // Center
                    if (f == d_alfa_0)
                        acc += d_r_d * w_ii_0 * w_jj_0;
                    if (f == d_alfa_1 + NUM_SECTOR)
                        acc += d_r_d * w_ii_0 * w_jj_0;

                    // Neighbor in Y
                    int ni = i + nearest_ii;
                    if (ni >= 0 && ni < sizeY) {
                        if (f == d_alfa_0)
                            acc += d_r_d * w_ii_1 * w_jj_0;
                        if (f == d_alfa_1 + NUM_SECTOR)
                            acc += d_r_d * w_ii_1 * w_jj_0;
                    }

                    // Neighbor in X
                    int nj = j + nearest_jj;
                    if (nj >= 0 && nj < sizeX) {
                        if (f == d_alfa_0)
                            acc += d_r_d * w_ii_0 * w_jj_1;
                        if (f == d_alfa_1 + NUM_SECTOR)
                            acc += d_r_d * w_ii_0 * w_jj_1;
                    }

                    // Neighbor in XY
                    if (ni >= 0 && ni < sizeY && nj >= 0 && nj < sizeX) {
                        if (f == d_alfa_0)
                            acc += d_r_d * w_ii_1 * w_jj_1;
                        if (f == d_alfa_1 + NUM_SECTOR)
                            acc += d_r_d * w_ii_1 * w_jj_1;
                    }
                }
            }
        }
    }

    __syncthreads();
    phase2 = (int)(clock64() - phase2);

    // Phase 3 Start: Write to global memory
    phase3 = clock64();
    if (i < sizeY && j < sizeX && f < numFeatures) {
        d_map[i * stringSize + j * numFeatures + f] = acc / (float)(k * k);  //FIX normalization
    }
    phase3 = (int)(clock64() - phase3);

    phase0 = clock64() - phase0;
/*
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 &&
        blockIdx.x == 0 && blockIdx.y == 0) {
        printf("Thread 0,0,0 of Block 0,0 took %d total cycles. It required:\n~ %d cycles to write to shared memory.\n~ %d cycles to compute data.\n~ %d cycles to convert shared memory to global memory.\n%d cycles unaccounted for.\n",
            (int)phase0, phase1, phase2, phase3, ((int)phase0 - (phase1 + phase2 + phase3)));
    }
*/
}

void featureGPU(int sizeY, int sizeX, int k, int height, int width, int numFeatures,
                float *map, int stringSize, int *alfa, float *r, float *w, int *nearest)
{
    float *d_map, *d_r, *d_w;
    int *d_alfa, *d_nearest;

    hipMalloc((void **)&d_map, sizeof(float) * (sizeX * sizeY * numFeatures));
    hipMalloc((void **)&d_alfa, sizeof(int) * (width * height * 2));
    hipMalloc((void **)&d_r, sizeof(float) * (width * height));
    hipMalloc((void **)&d_w, sizeof(float) * (k * 2));
    hipMalloc((void **)&d_nearest, sizeof(int) * k);

    hipMemcpy(d_map, map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyHostToDevice);
    hipMemcpy(d_alfa, alfa, sizeof(int) * (width * height * 2), hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(float) * (width * height), hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, sizeof(float) * (k * 2), hipMemcpyHostToDevice);
    hipMemcpy(d_nearest, nearest, sizeof(int) * k, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_X, BLOCK_Y, BLOCK_Z);
    dim3 blocksPerGrid(
        (sizeX + BLOCK_X - 1) / BLOCK_X,
        (sizeY + BLOCK_Y - 1) / BLOCK_Y);

    kernel_n4<<<blocksPerGrid, threadsPerBlock>>>(sizeY, sizeX, k, height, width, numFeatures,
                                                  d_map, stringSize, d_alfa, d_r, d_w, d_nearest);

    hipDeviceSynchronize(); // Ensure kernel completes before copying back

    hipMemcpy(map, d_map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyDeviceToHost);

    hipFree(d_map);
    hipFree(d_alfa);
    hipFree(d_r);
    hipFree(d_w);
    hipFree(d_nearest);
}
