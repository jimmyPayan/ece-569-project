#define NUM_SECTOR 9
#define BLOCK_DIM 8
#define K_MAX 4
#define FEATURES_MAX 81


#include <hip/hip_runtime.h>
#include <stdio.h>

// First optimization: Proper block dimensions (32x32 -> 16x16). Preparatory optimization that resulted in -10s from singlescale and -2s from Multiscale. Done so that shared memory (optimization 2) will not be > 48 kB.

// Still using atomics excessively, so we might not see speedup at all.  No removals of if() statements
// k seems to be passed in as cell_size, which is set to 4... for loop should be okay.

__global__ void kernel_n4(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *d_map, int stringSize, int *d_alfa, float *d_r, float *d_w,  int *d_nearest) {
	
	// Allocate shared memory
	__shared__ float shared_w[K_MAX * 2];
	__shared__ int shared_nearest[K_MAX];
	__shared__ float shared_blockMap[BLOCK_DIM * BLOCK_DIM * FEATURES_MAX];

	// Use thread IDs as iterators, same names as joaofaro to keep me sane while debugging
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int Idx = threadIdx.x + threadIdx.y * blockDim.x;	

	int a, d;

	if (Idx < k * 2)
		shared_w [Idx] = d_w[Idx];

	if (Idx < k)
		shared_nearest[Idx] = d_nearest[Idx];

	if (Idx < (BLOCK_DIM * BLOCK_DIM)) {
		for (a = 0 ; a < numFeatures; a++) {
			shared_blockMap[Idx * numFeatures + a] = 0.0f;
		}
	}

	__syncthreads();

if (i < sizeY && j < sizeX) {
    for (int ii = 0; ii < k; ii++) {
    for (int jj = 0; jj < k; jj++) {
        if ((i * k + ii > 0) && 
            (i * k + ii < height - 1) && 
            (j * k + jj > 0) && 
            (j * k + jj < width - 1))
        {
            d = (k * i + ii) * width + (j * k + jj);

            // d_map[ i * stringSize + j * numFeatures + d_alfa[d * 2    ]] += 
            //     d_r[d] * d_w[ii * 2] * d_w[jj * 2];
            shared_blockMap[Idx * numFeatures + d_alfa[d * 2]] += 
                d_r[d] * shared_w[ii * 2] * shared_w[jj * 2];

            // d_map[ i * stringSize + j * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
            //     d_r[d] * d_w[ii * 2] * d_w[jj * 2];
            shared_blockMap[Idx * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
                d_r[d] * shared_w[ii * 2] * shared_w[jj * 2];

            if ((threadIdx.x + shared_nearest[ii] >= 0) && 
                (threadIdx.x + shared_nearest[ii] < BLOCK_DIM))
            {

                // d_map[(i + nearest[ii]) * stringSize + j * numFeatures + d_alfa[d * 2    ]] += 
                //     d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2];
                shared_blockMap[((threadIdx.x + shared_nearest[ii]) + threadIdx.y * blockDim.x) * numFeatures + d_alfa[d * 2]] += 
                    d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2];

                // d_map[(i + nearest[ii]) * stringSize + j * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
                //     d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2];
                shared_blockMap[((threadIdx.x + shared_nearest[ii]) + threadIdx.y * blockDim.x) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
                    d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2];
            }

            if ((threadIdx.y + shared_nearest[jj] >= 0) && 
                (threadIdx.y + shared_nearest[jj] < BLOCK_DIM))
            {

                // d_map[i * stringSize + (j + nearest[jj]) * numFeatures + d_alfa[d * 2    ]] += 
                //     d_r[d] * d_w[ii * 2] * d_w[jj * 2 + 1];
                shared_blockMap[(threadIdx.x + (threadIdx.y + shared_nearest[jj]) * blockDim.x) * numFeatures + d_alfa[d * 2]] += 
                    d_r[d] * shared_w[ii * 2] * shared_w[jj * 2 + 1];

                // d_map[i * stringSize + (j + nearest[jj]) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
                //     d_r[d] * d_w[ii * 2] * d_w[jj * 2 + 1];
                shared_blockMap[(threadIdx.x + (threadIdx.y + shared_nearest[jj]) * blockDim.x) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
                    d_r[d] * shared_w[ii * 2] * shared_w[jj * 2 + 1];
            }

            if ((threadIdx.x + shared_nearest[ii] >= 0) && 
                (threadIdx.x + shared_nearest[ii] < BLOCK_DIM) &&
                (threadIdx.y + shared_nearest[jj] >= 0) && 
                (threadIdx.y + shared_nearest[jj] < BLOCK_DIM))
            {
 
                // d_map[(i + nearest[ii]) * stringSize + (j + nearest[jj]) * numFeatures + d_alfa[d * 2    ]] += 
                //     d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 + 1];
                shared_blockMap[((threadIdx.x + shared_nearest[ii]) + (threadIdx.y + shared_nearest[jj]) * blockDim.x) * numFeatures + d_alfa[d * 2]] += 
                    d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2 + 1];

                // d_map[(i + nearest[ii]) * stringSize + (j + nearest[jj]) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
                //     d_r[d] * d_w[ii * 2 + 1] * d_w[jj * 2 + 1];
                shared_blockMap[((threadIdx.x + shared_nearest[ii]) + (threadIdx.y + shared_nearest[jj]) * blockDim.x) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += 
                    d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2 + 1];
            }
        }
    }/*for(jj = 0; jj < k; jj++)*/
    }/*for(ii = 0; ii < k; ii++)*/
}/*if (i < sizeY && j < sizeX)*/


// Write shared memory to d_map
	if (i < sizeY && j < sizeX) {
    	for (int a = 0; a < numFeatures; a++) {
        	d_map[i * stringSize + j * numFeatures + a] = shared_blockMap[Idx * numFeatures + a];
    	}
	}

}


void featureGPU(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *map, int stringSize, int *alfa, float *r, float *w, int *nearest){
	//printf("Running kernel_n4.\n");
    float *d_map, *d_r, *d_w;
    int *d_alfa, *d_nearest;
// Commented out cudaMalloc()'s are passed by value
    //cudaMalloc((void**) &sizeY, sizeof(int));
    //cudaMalloc((void**) &sizeX, sizeof(int));
    //cudaMalloc((void**) &k,     sizeof(int));
    //cudaMalloc((void**) &height, sizeof(int);
    //cudaMalloc((void**) &width, sizeof(int));
    //cudaMalloc((void**) &numFeatures, sizeof(int));
    hipMalloc((void**) &d_map, sizeof(float) * (sizeX * sizeY * numFeatures));
    //cudaMalloc((void**) &stringSize, sizeof(int));
    hipMalloc((void**) &d_alfa, sizeof(int) * (width * height * 2));
    hipMalloc((void**) &d_r, sizeof(float) * (width * height));
    hipMalloc((void**) &d_w, sizeof(float) * (k * 2));
    hipMalloc((void**) &d_nearest, sizeof(int) * k);

    hipMemcpy(d_map, map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyHostToDevice);
    hipMemcpy(d_alfa, alfa, sizeof(int) * (width * height * 2) , hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(float) * (width * height), hipMemcpyHostToDevice); 
    hipMemcpy(d_w, w, sizeof(float) * (k * 2), hipMemcpyHostToDevice);
    hipMemcpy(d_nearest, nearest, sizeof(int) * k, hipMemcpyHostToDevice); 

    // Total number of threads needed: sizeY * sizeX * k... max value of k is 4, which occurs during HOG. 1024 / 4 = 256, sqrt(256) = 16.
    const dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);
    const dim3 blocksPerGrid(ceil((float)sizeY / BLOCK_DIM), ceil((float)sizeX / BLOCK_DIM));

    kernel_n4<<<blocksPerGrid,threadsPerBlock>>>(sizeY, sizeX, k, height, width, numFeatures, d_map, stringSize, d_alfa, d_r, d_w, d_nearest);

    hipMemcpy(map, d_map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyDeviceToHost);
    //cudaMemcpy(alfa, d_alfa, sizeof(int) * (width * height * 2) , cudaMemcpyDeviceToHost);
    //cudaMemcpy(r, d_r, sizeof(float) * (width * height), cudaMemcpyDeviceToHost); 
    //cudaMemcpy(w, d_w, sizeof(float) * (k * 2), cudaMemcpyDeviceToHost);
    //cudaMemcpy(nearest, d_nearest, sizeof(int) * k, cudaMemcpyDeviceToHost); 

    hipFree(d_map);
    hipFree(d_alfa);
    hipFree(d_r);
    hipFree(d_w);
    hipFree(d_nearest);
}
