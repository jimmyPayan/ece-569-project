#define NUM_SECTOR 9
#define BLOCK_DIM 16
#define K_MAX 4
#define FEATURES_MAX 27


#include <hip/hip_runtime.h>
#include <stdio.h>

// First optimization: Proper block dimensions (32x32 -> 16x16). Preparatory optimization that resulted in -10s from singlescale and -2s from Multiscale. Done so that shared memory (optimization 2) will not be > 48 kB.

// Still using atomics excessively, so we might not see speedup at all.  No removals of if() statements
// k seems to be passed in as cell_size, which is set to 4... for loop should be okay.

__global__ void kernel_n4(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *d_map, int stringSize, int *d_alfa, float *d_r, float *d_w,  int *d_nearest) {
	
	// Allocate shared memory
	__shared__ float shared_w[K_MAX * 2];
	__shared__ int shared_nearest[K_MAX];
	__shared__ float shared_blockMap[BLOCK_DIM * BLOCK_DIM * FEATURES_MAX];

	// Use thread IDs as iterators, same names as joaofaro to keep me sane while debugging
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int Idx = threadIdx.x + threadIdx.y * blockDim.x;	

	int a, d;

	int nearest_ii, nearest_jj;

	if (Idx < k * 2)
		shared_w [Idx] = d_w[Idx];

	if (Idx < k)
		shared_nearest[Idx] = d_nearest[Idx];

	if (Idx < (BLOCK_DIM * BLOCK_DIM)) {
		for (a = 0 ; a < numFeatures; a++) {
			shared_blockMap[Idx * numFeatures + a] = 0.0f;
		}
	}

	__syncthreads();

if (i < sizeY && j < sizeX) {
    for (int ii = 0; ii < k; ii++) {
    for (int jj = 0; jj < k; jj++) {
        if ((i * k + ii > 0) && 
            (i * k + ii < height - 1) && 
            (j * k + jj > 0) && 
            (j * k + jj < width - 1))
        {
            d = (k * i + ii) * width + (j * k + jj);
			
			nearest_ii = shared_nearest[ii];
			nearest_jj = shared_nearest[jj];

            shared_blockMap[Idx * numFeatures + d_alfa[d * 2]] += d_r[d] * shared_w[ii * 2] * shared_w[jj * 2];

            shared_blockMap[Idx * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += d_r[d] * shared_w[ii * 2] * shared_w[jj * 2];

			
            if (((int)threadIdx.x + nearest_ii >= 0) && 
                ((int)threadIdx.x + nearest_ii < BLOCK_DIM))
            {

                shared_blockMap[((threadIdx.x + nearest_ii) + threadIdx.y * blockDim.x) * numFeatures + d_alfa[d * 2]] += d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2];

                shared_blockMap[((threadIdx.x + nearest_ii) + threadIdx.y * blockDim.x) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2];
            }
			
			
            if (((int)threadIdx.y + nearest_jj >= 0) && 
                ((int)threadIdx.y + nearest_jj < BLOCK_DIM))
            {

                shared_blockMap[(threadIdx.x + (threadIdx.y + shared_nearest[jj]) * blockDim.x) * numFeatures + d_alfa[d * 2]] += d_r[d] * shared_w[ii * 2] * shared_w[jj * 2 + 1];

                shared_blockMap[(threadIdx.x + (threadIdx.y + shared_nearest[jj]) * blockDim.x) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += d_r[d] * shared_w[ii * 2] * shared_w[jj * 2 + 1];
            }

            if (((int)threadIdx.x + nearest_ii 	>= 0) && 
                ((int)threadIdx.x + nearest_ii 	< BLOCK_DIM) &&
                ((int)threadIdx.y + nearest_jj  >= 0) && 
                ((int)threadIdx.y + nearest_jj	< BLOCK_DIM))
            {

                shared_blockMap[((threadIdx.x + nearest_ii) + (threadIdx.y + nearest_jj) * blockDim.x) * numFeatures + d_alfa[d * 2]] += d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2 + 1];

                shared_blockMap[((threadIdx.x + nearest_ii) + (threadIdx.y + nearest_jj) * blockDim.x) * numFeatures + d_alfa[d * 2 + 1] + NUM_SECTOR] += d_r[d] * shared_w[ii * 2 + 1] * shared_w[jj * 2 + 1];
            }
        }
    }/*for(jj = 0; jj < k; jj++)*/
    }/*for(ii = 0; ii < k; ii++)*/
}/*if (i < sizeY && j < sizeX)*/

__syncthreads();

// Write to global memory
for (int a = 0; a < numFeatures; a++) {
    d_map[i * stringSize + j * numFeatures + a] = shared_blockMap[Idx * numFeatures + a];
}

}

void featureGPU(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *map, int stringSize, int *alfa, float *r, float *w, int *nearest){
	//printf("Running kernel_n4.\n");
    float *d_map, *d_r, *d_w;
    int *d_alfa, *d_nearest;
    hipMalloc((void**) &d_map, sizeof(float) * (sizeX * sizeY * numFeatures));
    hipMalloc((void**) &d_alfa, sizeof(int) * (width * height * 2));
    hipMalloc((void**) &d_r, sizeof(float) * (width * height));
    hipMalloc((void**) &d_w, sizeof(float) * (k * 2));
    hipMalloc((void**) &d_nearest, sizeof(int) * k);

    hipMemcpy(d_map, map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyHostToDevice);
    hipMemcpy(d_alfa, alfa, sizeof(int) * (width * height * 2) , hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(float) * (width * height), hipMemcpyHostToDevice); 
    hipMemcpy(d_w, w, sizeof(float) * (k * 2), hipMemcpyHostToDevice);
    hipMemcpy(d_nearest, nearest, sizeof(int) * k, hipMemcpyHostToDevice); 

    // Total number of threads needed: sizeY * sizeX * k... max value of k is 4, which occurs during HOG. 1024 / 4 = 256, sqrt(256) = 16.
    const dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);
    const dim3 blocksPerGrid(ceil((float)sizeY / BLOCK_DIM), ceil((float)sizeX / BLOCK_DIM));

    kernel_n4<<<blocksPerGrid,threadsPerBlock>>>(sizeY, sizeX, k, height, width, numFeatures, d_map, stringSize, d_alfa, d_r, d_w, d_nearest);

    hipMemcpy(map, d_map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyDeviceToHost);

    hipFree(d_map);
    hipFree(d_alfa);
    hipFree(d_r);
    hipFree(d_w);
    hipFree(d_nearest);
}
