#include "hip/hip_runtime.h"
#include "fhog.hpp"


#ifdef HAVE_TBB
#include <tbb/tbb.h>
#include "tbb/parallel_for.h"
#include "tbb/blocked_range.h"
#endif

#ifndef max
#define max(a,b)            (((a) > (b)) ? (a) : (b))
#endif

#ifndef min
#define min(a,b)            (((a) < (b)) ? (a) : (b))
#endif

#define NUM_SECTOR 9

// Naive 3D Approach first. Can do pseudo-4D if needed. No memory optimizations yet, no removals of if() statements
// k seems to be passed in as cell_size, which is set to 4... for loop should be okay.

__globabl__ void kernel_n4(int sizeY, int sizeX, int k, int height, int width, float *map, int stringSize, int *alfa, float *r, float *w,  int *nearest) {
	
	// Use thread IDs as iterators, same names as joaofaro to keep me sane while debugging
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int ii = blockIdx.z * blockDim.z + threadIdx.z;
	
	int d;
if (i < sizeY && j < sizeX) {
	for(int jj = 0; jj < k; jj++)
          {
            if ((i * k + ii > 0) && 
                (i * k + ii < height - 1) && 
                (j * k + jj > 0) && 
                (j * k + jj < width  - 1))
            {
              d = (k * i + ii) * width + (j * k + jj);
              map[ i * stringSize + j * (*map)->numFeatures + alfa[d * 2    ]] += 
                  r[d] * w[ii * 2] * w[jj * 2];
              map[ i * stringSize + j * (*map)->numFeatures + alfa[d * 2 + 1] + NUM_SECTOR] += 
                  r[d] * w[ii * 2] * w[jj * 2];
              if ((i + nearest[ii] >= 0) && 
                  (i + nearest[ii] <= sizeY - 1))
              {
                map[(i + nearest[ii]) * stringSize + j * (*map)->numFeatures + alfa[d * 2    ]             ] += 
                  r[d] * w[ii * 2 + 1] * w[jj * 2 ];
                map[(i + nearest[ii]) * stringSize + j * (*map)->numFeatures + alfa[d * 2 + 1] + NUM_SECTOR] += 
                  r[d] * w[ii * 2 + 1] * w[jj * 2 ];
              }
              if ((j + nearest[jj] >= 0) && 
                  (j + nearest[jj] <= sizeX - 1))
              {
                map[i * stringSize + (j + nearest[jj]) * (*map)->numFeatures + alfa[d * 2    ]             ] += 
                  r[d] * w[ii * 2] * w[jj * 2 + 1];
                map[i * stringSize + (j + nearest[jj]) * (*map)->numFeatures + alfa[d * 2 + 1] + NUM_SECTOR] += 
                  r[d] * w[ii * 2] * w[jj * 2 + 1];
              }
              if ((i + nearest[ii] >= 0) && 
                  (i + nearest[ii] <= sizeY - 1) && 
                  (j + nearest[jj] >= 0) && 
                  (j + nearest[jj] <= sizeX - 1))
              {
                map[(i + nearest[ii]) * stringSize + (j + nearest[jj]) * (*map)->numFeatures + alfa[d * 2    ]             ] += 
                  r[d] * w[ii * 2 + 1] * w[jj * 2 + 1];
                map[(i + nearest[ii]) * stringSize + (j + nearest[jj]) * (*map)->numFeatures + alfa[d * 2 + 1] + NUM_SECTOR] += 
                  r[d] * w[ii * 2 + 1] * w[jj * 2 + 1];
              }
            } // for(int jj = 0; jj < k; jj++)
} // if (i < sizeY && j < sizeX)

}
