#define NUM_SECTOR 9
#define BLOCK_DIM 1
#define K_MAX 4
#define FEATURES_MAX 27


#include <hip/hip_runtime.h>
#include <stdio.h>

// Timing functions  give tools to directly evaluate speedups! Comment out if you want max performance, I suppose.

/*  NOTE: Clock Rate = 1328500 kHz */

/* 	UPDATE THIS COMMENT AFTER ANY AND ALL OPTIMIZATIONS PLEASE <3 
	As a convention, just use the last thread measurement and copy alongside the Function Timing Summary
		

	OPTIMIZATION NAME: 27x1 Coalesced Memory with minimal global reads.

	MULTISCALE
	Thread 0,0 of Block 0,0 took 6560 total cycles. It required:
	~ 671 cycles to write to shared memory.
	~ 5387 cycles to compute data.
	~ 176 cycles to convert shared memory to global memory.
	326 cycles unaccounted for.
	Total execution time: 18470.0 ms.

	--- Function Timing Summary ---
	Total time spent in getFeatures(): 2.4 s
	Total time spent in gaussianCorrelation(): 0.7 s
	Total time spent in train(): -2.2 s
	Total time spent in detect(): -1.9 s
	Total execution time: -1.0 s


	SINGLESCALE
	Thread 0,0 of Block 0,0 took 10005 total cycles. It required:
	~ 882 cycles to write to shared memory.
	~ 8573 cycles to compute data.
	~ 222 cycles to convert shared memory to global memory.
	328 cycles unaccounted for.
	Total execution time: 103860.0 ms.

	--- Function Timing Summary ---
	Total time spent in getFeatures(): 23.3 s
	Total time spent in gaussianCorrelation(): 74.8 s
	Total time spent in train(): 14.9 s
	Total time spent in detect(): 14.7 s
	Total execution time: 127.7 s



*/


// No removals of if() statements

__global__ void kernel_n4(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *d_map, int stringSize, int *d_alfa, float *d_r, float *d_w, int *d_nearest) {
	// Phase 0 Start: Overall thread duration
	long long int phase0 = clock64();

	__shared__ float shared_w[K_MAX * 2];
	__shared__ int shared_nearest[K_MAX];
	__shared__ float shared_blockMap[FEATURES_MAX];

	int i = blockIdx.y;
	int j = blockIdx.x;
	int featureIdx = threadIdx.x;  // 0..26

	int phase1, phase2, phase3;

	int d;

	// Local variables for data accessed multiple times
	int nearest_ii, nearest_jj;
	int d_alfa_0, d_alfa_1;
	float w_ii_0, w_ii_1, w_jj_0, w_jj_1;
	float d_r_d;

	// Phase 1 Start: Initialize shared memory
	phase1 = clock64();
	if (featureIdx < k * 2) 
		shared_w[featureIdx] = d_w[featureIdx];

	if (featureIdx < k) 
		shared_nearest[featureIdx] = d_nearest[featureIdx];

	if (featureIdx < numFeatures)
		shared_blockMap[featureIdx] = 0.0f;

	__syncthreads();
	// Phase 1 End: Initialize shared memory
	phase1 = (int)clock64() - phase1;

	// One thread per feature, one block per cell
	// Phase 2 Start: Compute
	phase2 = clock64();
	if (featureIdx == 0) {

		if (i < sizeY && j < sizeX) {
			for (int ii = 0; ii < k; ii++) {
				for (int jj = 0; jj < k; jj++) {
					if ((i * k + ii > 0) && (i * k + ii < height - 1) &&
						(j * k + jj > 0) && (j * k + jj < width  - 1))
					{
						d = (k * i + ii) * width + (j * k + jj);

						nearest_ii = shared_nearest[ii];
						nearest_jj = shared_nearest[jj];
						d_alfa_0 = d_alfa[d * 2];			
						d_alfa_1 = d_alfa[d * 2 + 1];	
						w_ii_0 = shared_w[ii * 2];
						w_ii_1 = shared_w[ii * 2 + 1];
						w_jj_0 = shared_w[jj * 2];
						w_jj_1 = shared_w[jj * 2 + 1];
						d_r_d = d_r[d];

						shared_blockMap[d_alfa_0] += d_r_d * w_ii_0 * w_jj_0;
						shared_blockMap[d_alfa_1 + NUM_SECTOR] += d_r_d * w_ii_0 * w_jj_0;

						if ((i + nearest_ii >= 0) && (i + nearest_ii < sizeY)) {
							shared_blockMap[d_alfa_0] += d_r_d * w_jj_1 * w_jj_0;
							shared_blockMap[d_alfa_1 + NUM_SECTOR] += d_r_d * w_jj_1 * w_jj_0;
						}

						if ((j + nearest_jj >= 0) && (j + nearest_jj < sizeX)) {
							shared_blockMap[d_alfa_0] += d_r_d * w_ii_0 * w_jj_1;
							shared_blockMap[d_alfa_1 + NUM_SECTOR] += d_r_d * w_ii_0 * w_jj_1;
						}

						if ((i + nearest_ii >= 0) && (i + nearest_ii < sizeY) &&
							(j + nearest_jj >= 0) && (j + nearest_jj < sizeX)) {
							shared_blockMap[d_alfa_0] += d_r_d * w_ii_1 * w_jj_1;
							shared_blockMap[d_alfa_1 + NUM_SECTOR] += d_r_d * w_ii_1 * w_jj_1;
						}
					}
				}
			}
		}
	}

	__syncthreads();
	// Phase 2 End: Compute
	phase2 = (int)clock64() - phase2;


	// Phase 3 Start: Write to global memory
	phase3 = clock64();
	if (i < sizeY && j < sizeX && featureIdx < numFeatures) {
		d_map[i * stringSize + j * numFeatures + featureIdx] = shared_blockMap[featureIdx];
	}
	// Phase 3 End: Write to global memory
	phase3 = (int)clock64() - phase3;

	//Phase 0 End: Overall thread duration
	phase0 = clock64() - phase0;

	if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
		printf("Thread 0,0 of Block 0,0 took %d total cycles. It required:\n~ %d cycles to write to shared memory.\n~ %d cycles to compute data.\n~ %d cycles to convert shared memory to global memory.\n%d cycles unaccounted for.\n", (int) phase0, phase1, phase2, phase3, ((int) phase0 - (phase1 + phase2 + phase3)));	
	}
	
}



void featureGPU(int sizeY, int sizeX, int k, int height, int width, int numFeatures, float *map, int stringSize, int *alfa, float *r, float *w, int *nearest){
	/*
	cudaDeviceProp prop;
	cudaGetDeviceProperties(&prop, 0); // 0 = device ID (first GPU)
	printf("Clock Rate: %d kHz\n", (int)prop.clockRate);
	*/  

	float *d_map, *d_r, *d_w;
    int *d_alfa, *d_nearest;
    hipMalloc((void**) &d_map, sizeof(float) * (sizeX * sizeY * numFeatures));
    hipMalloc((void**) &d_alfa, sizeof(int) * (width * height * 2));
    hipMalloc((void**) &d_r, sizeof(float) * (width * height));
    hipMalloc((void**) &d_w, sizeof(float) * (k * 2));
    hipMalloc((void**) &d_nearest, sizeof(int) * k);

    hipMemcpy(d_map, map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyHostToDevice);
    hipMemcpy(d_alfa, alfa, sizeof(int) * (width * height * 2) , hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(float) * (width * height), hipMemcpyHostToDevice); 
    hipMemcpy(d_w, w, sizeof(float) * (k * 2), hipMemcpyHostToDevice);
    hipMemcpy(d_nearest, nearest, sizeof(int) * k, hipMemcpyHostToDevice); 

    const dim3 threadsPerBlock(FEATURES_MAX, 1);
    const dim3 blocksPerGrid(sizeX, sizeY);

    kernel_n4<<<blocksPerGrid,threadsPerBlock>>>(sizeY, sizeX, k, height, width, numFeatures, d_map, stringSize, d_alfa, d_r, d_w, d_nearest);

    hipMemcpy(map, d_map, sizeof(float) * (sizeX * sizeY * numFeatures), hipMemcpyDeviceToHost);

    hipFree(d_map);
    hipFree(d_alfa);
    hipFree(d_r);
    hipFree(d_w);
    hipFree(d_nearest);
}

