#include "hip/hip_runtime.h"
// Ryan Raad 2025
// normalizeAndTruncate.cu
// most naive Cuda implementation, single pass, single kernel, worst speedup

#include "normalizeAndTruncate.cuh"
#include "fhog.hpp"           // for CvLSVMFeatureMapCaskade, NUM_SECTOR, LATENT_SVM_OK
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>             // for printf
#include <cstdlib>            // for malloc/free

#define CUDA_CHECK(err) \
  if ((err) != hipSuccess) { \
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err)); \
    exit(EXIT_FAILURE); \
  }

// -----------------------------------------------------------------------------
//  Kernel: normalizeAndTruncateNaiveKernel
// -----------------------------------------------------------------------------
__global__ void normalizeAndTruncateNaiveKernel(
    const float* __restrict__ mapData,
    float*       __restrict__ outData,
    int oldSizeX, int oldSizeY,
    float alfa)
{
    int idx       = blockIdx.x*blockDim.x + threadIdx.x;
    int newSizeX  = oldSizeX - 2;
    int newSizeY  = oldSizeY - 2;
    int cells     = newSizeX * newSizeY;
    const int p   = NUM_SECTOR;
    const int xp  = NUM_SECTOR*3;
    const int pp  = NUM_SECTOR*12;

    if (idx < cells) {
        int j = (idx % newSizeX) + 1;   // 1…oldSizeX–2
        int i = (idx / newSizeX) + 1;   // 1…oldSizeY–2

        // compute the four cell‐norms on the fly
        float sum0=0, sum1=0, sum2=0, sum3=0;
        int offsets[4] = {
          ( i   * oldSizeX +  j   )*xp,
          ( i   * oldSizeX + (j+1))*xp,
          ((i+1)* oldSizeX +  j   )*xp,
          ((i+1)* oldSizeX + (j+1))*xp
        };
        for (int c=0; c<p; ++c){
          float v = mapData[offsets[0]+c]; sum0 += v*v;
          v       = mapData[offsets[1]+c]; sum1 += v*v;
          v       = mapData[offsets[2]+c]; sum2 += v*v;
          v       = mapData[offsets[3]+c]; sum3 += v*v;
        }
        float norm = sqrtf(sum0+sum1+sum2+sum3 + 1e-6f);

        // normalize & clamp
        int outBase = idx * pp;
        for (int c=0; c<pp; ++c) {
          float val = mapData[offsets[0] + c] / norm;
          outData[outBase + c] = (val > alfa ? alfa : val);
        }
    }
}

// -----------------------------------------------------------------------------
//  Host wrapper: normalizeAndTruncateNaive()
// -----------------------------------------------------------------------------
int normalizeAndTruncateNaive(CvLSVMFeatureMapCaskade* map, float alfa)
{
    // 1) Gather sizes
    int oldSizeX   = map->sizeX;
    int oldSizeY   = map->sizeY;
    const int p    = NUM_SECTOR;
    const int xp   = p*3;
    const int pp   = p*12;
    int totalCells = oldSizeX * oldSizeY;
    int newSizeX   = oldSizeX - 2;
    int newSizeY   = oldSizeY - 2;
    int newCells   = newSizeX * newSizeY;

    size_t mapBytes = sizeof(float) * totalCells * xp;
    size_t outBytes = sizeof(float) * newCells   * pp;

    // 2) Allocate & copy input map to device
    float *d_map, *d_out;
    CUDA_CHECK(hipMalloc(&d_map, mapBytes));
    CUDA_CHECK(hipMemcpy(d_map, map->map, mapBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc(&d_out, outBytes));

    // 3) Launch kernel
    int threads = 256;
    int blocks1 = (newCells + threads - 1) / threads;
    normalizeAndTruncateNaiveKernel<<<blocks1, threads>>>(d_map, d_out, oldSizeX, oldSizeY, alfa);
    CUDA_CHECK(hipDeviceSynchronize());

    // 4) Copy result back
    float* h_out = (float*)malloc(outBytes);
    CUDA_CHECK(hipMemcpy(h_out, d_out, outBytes, hipMemcpyDeviceToHost));

    // 5) Clean up device
    hipFree(d_map);
    hipFree(d_out);

    // 6) Replace host map with new data
    free(map->map);
    map->map         = h_out;
    map->sizeX       = newSizeX;
    map->sizeY       = newSizeY;
    map->numFeatures = pp;

    return LATENT_SVM_OK;
}
