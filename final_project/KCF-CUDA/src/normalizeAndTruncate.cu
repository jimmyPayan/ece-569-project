#include "hip/hip_runtime.h"
// Ryan Raad 2025
// normalizeAndTruncate.cu
// most naive Cuda implementation, single pass, single kernel, worst speedup
#include "normalizeAndTruncate.cuh"
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void normalizeAndTruncateNaive(
    const float* __restrict__ mapData,
    float*       __restrict__ outData,
    int oldSizeX, int oldSizeY,
    float alfa)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int newSizeX = oldSizeX - 2;
    int newSizeY = oldSizeY - 2;
    int cells = newSizeX * newSizeY;
    const int p  = NUM_SECTOR;
    const int xp = NUM_SECTOR*3;
    const int pp = NUM_SECTOR*12;

    if (idx < cells) {
        int j = (idx % newSizeX) + 1;   // 1…oldSizeX–2
        int i = (idx / newSizeX) + 1;   // 1…oldSizeY–2

        // 1) on-the-fly compute the norm of the 4 cells
        float sum0=0, sum1=0, sum2=0, sum3=0;
        int offsets[4] = {
          ( i   * oldSizeX +  j   )*xp,
          ( i   * oldSizeX + (j+1))*xp,
          ((i+1)* oldSizeX +  j   )*xp,
          ((i+1)* oldSizeX + (j+1))*xp
        };
        for (int c=0; c<p; ++c){
          float v;
          v = mapData[offsets[0]+c]; sum0 += v*v;
          v = mapData[offsets[1]+c]; sum1 += v*v;
          v = mapData[offsets[2]+c]; sum2 += v*v;
          v = mapData[offsets[3]+c]; sum3 += v*v;
        }
        float norm = sqrtf(sum0+sum1+sum2+sum3 + 1e-6f);

        // 2) normalize & clamp each of the pp features
        int outBase = idx * pp;
        for (int c=0; c<pp; ++c) {
          // pick corresponding mapData element (this is simplified—your real layout may vary)
          float val = mapData[offsets[0] + c];  
          val = val / norm;
          outData[outBase + c] = (val > alfa ? alfa : val);
        }
    }
}

// Host wrapper omitted for brevity—just hipMalloc/copy, launch this kernel, copy back.
